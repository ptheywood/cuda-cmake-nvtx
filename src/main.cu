#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "nvtx_util.cuh"

static void HandleCUDAError(const char *file, int line, hipError_t status = hipGetLastError()) {
	if (status != hipSuccess || (status = hipGetLastError()) != hipSuccess)
	{
		if (status == hipErrorUnknown)
		{
			printf("%s(%i) An Unknown CUDA Error Occurred :(\n", file, line);
			exit(1);
		}
		printf("%s(%i) CUDA Error Occurred;\n%s\n", file, line, hipGetErrorString(status));
		exit(1);
	}
}

#define CUDA_CALL( err ) (HandleCUDAError(__FILE__, __LINE__ , err))
#define CUDA_CHECK() (HandleCUDAError(__FILE__, __LINE__))



__global__ void pow2Kernel(const unsigned int N, const unsigned int reps, unsigned int * in, unsigned int * out){
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < N){
        for(unsigned int rep = 0; rep < reps; rep++){
            out[idx] = in[idx] * in[idx];
        }
    }
}

void pow2Host(const unsigned int N, const unsigned int reps, unsigned int * in, unsigned int * out){
    for(unsigned int idx = 0; idx < N; idx++){
        for(unsigned int rep = 0; rep < reps; rep++){
            out[idx] = in[idx] * in[idx];
        }
    }
}

unsigned int evaluate(const unsigned int N, unsigned int * a, unsigned int * b, const unsigned int print_count){
    NVTX_RANGE("evaluate");
    unsigned int difference_count = 0;
    for(unsigned int idx = 0; idx < N; idx++){
        bool match = a[idx] == b[idx];
        if(!match){
            difference_count++;
        }
        if(idx < print_count){
            printf("%u: %u == %u ? %d\n", idx, a[idx], b[idx], match);
        }
    }
    return difference_count;
}

bool allocate(const unsigned int N, unsigned int ** h_in, unsigned int ** h_out, unsigned int ** d_in, unsigned int ** d_out, unsigned int ** h_d_out){
    NVTX_RANGE("allocate");
    size_t size = N * sizeof(unsigned int);

    *h_in = (unsigned int*) malloc(size);
    *h_out = (unsigned int*) malloc(size);
    CUDA_CALL(hipMalloc((void**)d_in, size));
    CUDA_CALL(hipMalloc((void**)d_out, size));
    *h_d_out = (unsigned int*) malloc(size);

    bool success = true;
    if(*h_in == nullptr){
        success = false;
    }
    if(*h_out == nullptr){
        success = false;
    }
    if(*h_d_out == nullptr){
        success = false;
    }
    return success;
}

void initialse(const unsigned int N, unsigned int * h_in, unsigned int * h_out, unsigned int * d_in, unsigned int * d_out, unsigned int * h_d_out){
    NVTX_RANGE("initialse");

    size_t size = N * sizeof(unsigned int);

    // Set memory values to 0
    memset(h_in, 0, size);
    memset(h_out, 0, size);
    CUDA_CALL(hipMemset(d_in, 0, size));
    CUDA_CALL(hipMemset(d_out, 0, size));
    memset(h_d_out, 0, size);

    // Initialise the host input.
    for(unsigned int idx = 0; idx < N; idx++){
        h_in[idx] = idx;
    }
}
void executeOnDevice(const unsigned int N, const unsigned int reps, unsigned int * h_in, unsigned int * d_in, unsigned int * d_out, unsigned int * h_d_out){
    NVTX_RANGE("executeOnDevice");
    const unsigned int kernel_reps = 32;
    size_t size = N * sizeof(unsigned int);

    NVTX_PUSH("H2D");
    // Copy input to device
    CUDA_CALL(hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice));
    NVTX_POP();
    // Launch kernel
    int blockSize = 0;
	int minGridSize = 0;
	int gridSize = 0;
    CUDA_CALL(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, pow2Kernel, 0, N));
    gridSize = (N + blockSize - 1) / blockSize;

    NVTX_PUSH("kerenel_reps");
    for(unsigned int krep = 0; krep < kernel_reps; krep++){
        NVTX_PUSH("pow2Kernel");
        pow2Kernel << <gridSize, blockSize >> >(N, reps, d_in, d_out);
        NVTX_POP();
    }
    hipDeviceSynchronize();
    CUDA_CHECK();
    NVTX_POP();

    
    // Copy data from device to host.
    NVTX_PUSH("D2H");
    CUDA_CALL(hipMemcpy(h_d_out, d_out, size, hipMemcpyDeviceToHost));
    NVTX_POP();
}

void executeOnHost(const unsigned int N, const unsigned int reps, unsigned int * h_in, unsigned int * h_out){
    NVTX_RANGE("executeOnHost");
    pow2Host(N, reps, h_in, h_out);
}

void deallocate(unsigned int ** h_in, unsigned int ** h_out, unsigned int ** d_in, unsigned int ** d_out, unsigned int ** h_d_out){
    NVTX_RANGE("deallocate");
    free(*h_in);
    *h_in = nullptr;
    free(*h_out);
    *h_out = nullptr;
    
    CUDA_CALL(hipFree(*d_in));
    *d_in = nullptr;
    CUDA_CALL(hipFree(*d_out));
    *d_out = nullptr;
    
    free(*h_d_out);
    *h_d_out = nullptr;
}

bool arbitraryCUDAStuff(){
    // Push a range marker.
    NVTX_RANGE("arbitraryCUDAStuff");

    // Set problem size
    // const unsigned int N = 1024;
    const unsigned int N = 65536;
    const unsigned int reps = 32;

    // Declare pointers
    unsigned int * h_in = nullptr;
    unsigned int * h_out = nullptr;
    unsigned int * d_in = nullptr;
    unsigned int * d_out = nullptr;
    unsigned int * h_d_out = nullptr;

    // Allocate
    bool allocated = allocate(N, &h_in, &h_out, &d_in, &d_out, &h_d_out);
    if(!allocated){
        return false;
    }

    // Initialise
    initialse(N, h_in, h_out, d_in, d_out, h_d_out);
    
    // Execute Device
    executeOnDevice(N, reps, h_in, d_in, d_out, h_d_out);

    // Execute host
    executeOnHost(N, reps, h_in, h_out);

    // Evalute
    const unsigned int print_count = 0;
    unsigned int error_count = evaluate(N, h_out, h_d_out, print_count);
    if(error_count != 0){
        printf("Incorrect: %u incorrect values\n", error_count);
    } else {
        printf("Success!\n");
    }

    // Free 
    deallocate(&h_in, &h_out, &d_in, &d_out, &h_d_out);

    return !error_count;
}

void cudaInit(){
    NVTX_RANGE("cudaInit");
    // Free the nullptr to initialise the cuda context.
    CUDA_CALL(hipFree(0));
}

void printNVTXStatus(){
    #if defined(USE_NVTX)
        printf("NVTX is ON\n");
    #else 
        printf("NVTX is OFF\n");
    #endif
}

int main(int argc, char * argv[]){
    // Print if NVTX is enabled or not.
    printNVTXStatus();

    // Explicit full main markers.
    NVTX_PUSH("main");

    // Early initialise the cuda context to improve profiling clarity.
    cudaInit();

    // Run some stuff.
    bool success = arbitraryCUDAStuff();

    NVTX_POP();

    // Reset the device.
    hipDeviceReset();

    return success ? EXIT_SUCCESS : EXIT_FAILURE;
}
